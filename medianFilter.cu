#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <iostream>
#include "MedianFilter.h"
#include <time.h>
#define TILE_SIZE 4 

__global__ void medianFilterKernel(unsigned char *inputImageKernel, unsigned char *outputImagekernel, int imageWidth, int imageHeight)
{
	// Set row and colum for thread.
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned char filterVector[9] = {0,0,0,0,0,0,0,0,0};   //Take fiter window
	if((row==0) || (col==0) || (row==imageHeight-1) || (col==imageWidth-1))
				outputImagekernel[row*imageWidth+col] = 0; //Deal with boundry conditions
	else {
		for (int x = 0; x < WINDOW_SIZE; x++) { 
			for (int y = 0; y < WINDOW_SIZE; y++){
				filterVector[x*WINDOW_SIZE+y] = inputImageKernel[(row+x-1)*imageWidth+(col+y-1)];   // setup the filterign window.
			}
		}
		for (int i = 0; i < 9; i++) {
			for (int j = i + 1; j < 9; j++) {
				if (filterVector[i] > filterVector[j]) { 
					//Swap the variables.
					char tmp = filterVector[i];
					filterVector[i] = filterVector[j];
					filterVector[j] = tmp;
				}
			}
		}
		outputImagekernel[row*imageWidth+col] = filterVector[4];   //Set the output variables.
	}
}


__global__ void medianFilterSharedKernel(unsigned char *inputImageKernel, unsigned char *outputImagekernel, int imageWidth, int imageHeight)
{
	//Set the row and col value for each thread.
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ unsigned char sharedmem[(TILE_SIZE+2)]  [(TILE_SIZE+2)];  //initialize shared memory
	//Take some values.
	bool is_x_left = (threadIdx.x == 0), is_x_right = (threadIdx.x == TILE_SIZE-1);
    bool is_y_top = (threadIdx.y == 0), is_y_bottom = (threadIdx.y == TILE_SIZE-1);

	//Initialize with zero
	if(is_x_left)
		sharedmem[threadIdx.x][threadIdx.y+1] = 0;
	else if(is_x_right)
		sharedmem[threadIdx.x + 2][threadIdx.y+1]=0;
	if (is_y_top){
		sharedmem[threadIdx.x+1][threadIdx.y] = 0;
		if(is_x_left)
			sharedmem[threadIdx.x][threadIdx.y] = 0;
		else if(is_x_right)
			sharedmem[threadIdx.x+2][threadIdx.y] = 0;
	}
	else if (is_y_bottom){
		sharedmem[threadIdx.x+1][threadIdx.y+2] = 0;
		if(is_x_right)
			sharedmem[threadIdx.x+2][threadIdx.y+2] = 0;
		else if(is_x_left)
			sharedmem[threadIdx.x][threadIdx.y+2] = 0;
	}

	//Setup pixel values
	sharedmem[threadIdx.x+1][threadIdx.y+1] = inputImageKernel[row*imageWidth+col];
	//Check for boundry conditions.
	if(is_x_left && (col>0))
		sharedmem[threadIdx.x][threadIdx.y+1] = inputImageKernel[row*imageWidth+(col-1)];
	else if(is_x_right && (col<imageWidth-1))
		sharedmem[threadIdx.x + 2][threadIdx.y+1]= inputImageKernel[row*imageWidth+(col+1)];
	if (is_y_top && (row>0)){
		sharedmem[threadIdx.x+1][threadIdx.y] = inputImageKernel[(row-1)*imageWidth+col];
		if(is_x_left)
			sharedmem[threadIdx.x][threadIdx.y] = inputImageKernel[(row-1)*imageWidth+(col-1)];
		else if(is_x_right )
			sharedmem[threadIdx.x+2][threadIdx.y] = inputImageKernel[(row-1)*imageWidth+(col+1)];
	}
	else if (is_y_bottom && (row<imageHeight-1)){
		sharedmem[threadIdx.x+1][threadIdx.y+2] = inputImageKernel[(row+1)*imageWidth + col];
		if(is_x_right)
			sharedmem[threadIdx.x+2][threadIdx.y+2] = inputImageKernel[(row+1)*imageWidth+(col+1)];
		else if(is_x_left)
			sharedmem[threadIdx.x][threadIdx.y+2] = inputImageKernel[(row+1)*imageWidth+(col-1)];
	}

	__syncthreads();   //Wait for all threads to be done.

	//Setup the filter.
	unsigned char filterVector[9] = {sharedmem[threadIdx.x][threadIdx.y], sharedmem[threadIdx.x+1][threadIdx.y], sharedmem[threadIdx.x+2][threadIdx.y],
                   sharedmem[threadIdx.x][threadIdx.y+1], sharedmem[threadIdx.x+1][threadIdx.y+1], sharedmem[threadIdx.x+2][threadIdx.y+1],
                   sharedmem[threadIdx.x] [threadIdx.y+2], sharedmem[threadIdx.x+1][threadIdx.y+2], sharedmem[threadIdx.x+2][threadIdx.y+2]};

	
	{
		for (int i = 0; i < 9; i++) {
        for (int j = i + 1; j < 9; j++) {
            if (filterVector[i] > filterVector[j]) { 
				//Swap Values.
                char tmp = filterVector[i];
                filterVector[i] = filterVector[j];
                filterVector[j] = tmp;
            }
        }
    }
	outputImagekernel[row*imageWidth+col] = filterVector[4];   //Set the output image values.
	}
}

bool MedianFilterGPU( Bitmap* image, Bitmap* outputImage, bool sharedMemoryUse ){
	//Cuda error and image values.
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	hipError_t status;
	int width = image->Width();
	int height = image->Height();

	int size =  width * height * sizeof(char);
	//initialize images.
	unsigned char *deviceinputimage;
	hipMalloc((void**) &deviceinputimage, size);
	status = hipGetLastError();              
	if (status != hipSuccess) {                     
		std::cout << "Kernel failed for hipMalloc : " << hipGetErrorString(status) << 
		std::endl;
		return false;
	}
	hipMemcpy(deviceinputimage, image->image, size, hipMemcpyHostToDevice);
	status = hipGetLastError();              
	if (status != hipSuccess) {                     
		std::cout << "Kernel failed for hipMemcpy hipMemcpyHostToDevice: " << hipGetErrorString(status) << 
		std::endl;
		hipFree(deviceinputimage);
		return false;
	}
	unsigned char *deviceOutputImage;
	hipMalloc((void**) &deviceOutputImage, size);
	//take block and grids.
	dim3 dimBlock(TILE_SIZE, TILE_SIZE);
	dim3 dimGrid((int)ceil((float)image->Width() / (float)TILE_SIZE),
				(int)ceil((float)image->Height() / (float)TILE_SIZE));

	//Check for shared memories and call the kernel
	if (!sharedMemoryUse)
		medianFilterKernel<<<dimGrid, dimBlock>>>(deviceinputimage, deviceOutputImage, width, height);
	else
		medianFilterSharedKernel<<<dimGrid, dimBlock>>>(deviceinputimage, deviceOutputImage, width, height);
	
	

// save output image to host.
	hipMemcpy(outputImage->image, deviceOutputImage, size, hipMemcpyDeviceToHost);
	status = hipGetLastError();              
	


if (status != hipSuccess) {                     
		std::cout << "Kernel failed for hipMemcpy hipMemcpyDeviceToHost: " << hipGetErrorString(status) << 
		std::endl;
		hipFree(deviceinputimage);
		hipFree(deviceOutputImage);
		return false;
	}
	//Free the memory
	hipFree(deviceinputimage);
	hipFree(deviceOutputImage);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float time = 0;
	hipEventElapsedTime(&time,start,stop);
	printf("time %f\n", time); 
	return true;
}
